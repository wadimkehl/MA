#include "hip/hip_runtime.h"


#include "gpu.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include "sys/time.h"

using namespace std;

#define WIN_SIZE 0


#define RED(x)   ((x >> 16) & 0xff)
#define GREEN(x) ((x >> 8) & 0xff)
#define BLUE(x)  (x & 0xff)

#define SQRT_2PI 2.50662827f


__global__ void kernel_segmentation(float *likely,float *primal, float *dual, float *tmp, float *g, float tau, int nx, int ny, int nr_labels,
                                    size_t lt_p,size_t pd_p, size_t g_p, int method)
{

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int pos = y*pd_p + x, c = pd_p*ny, page=2*c, book=pd_p*ny*nr_labels;

    bool work = x < nx && y < ny;
    work = (x >= WIN_SIZE && x < nx-WIN_SIZE && y >= WIN_SIZE  && y < ny-WIN_SIZE);

    //primal holds dashed V_n+1
    //primal[book] holds normal primal variable V_n+1

    // 1st step
    if (work)
    {

        // Add tau*grad to dual and consider boundaries
        if(x<nx-1-WIN_SIZE)
            for(int l=0; l < nr_labels;l++)
                dual[l*page     +pos] += tau*(primal[l*c +pos  +1]-primal[l*c + pos]);

        if(y<ny-1-WIN_SIZE)
            for(int l=0; l < nr_labels;l++)
                dual[l*page + c +pos] += tau*(primal[l*c +pos +pd_p]-primal[l*c + pos]);

        if (method==0)
        {
            //Lellmann dual space
            float v=0;
            for(unsigned int i = 0; i < nr_labels; i++)
                v += dual[pos     + page*i]*dual[pos     + page*i]+dual[pos + c + page*i]*dual[pos + c + page*i];
            if (v>1) //if (v>g[y*g_p + x])
            {
                v = sqrt(v);
                for(unsigned int i = 0; i < nr_labels; i++)
                {
                    dual[pos     + page*i] /= v;
                    dual[pos + c + page*i] /= v;
                }
            }
        }
        else if (method==1)
        {
            // Zach dual space
            float v=0;
            for(unsigned int i = 0; i < nr_labels; i++)
                v += abs(dual[pos     + page*i])+abs(dual[pos + c + page*i]);
            if (v>1.0f)
                for(unsigned int i = 0; i < nr_labels; i++)
                {
                    dual[pos     + page*i] /= v;
                    dual[pos + c + page*i] /= v;
                }
        }

        else
        {

            //      Chambolle dual space
            // Do Dykstra projection
            float diffus=g[y*g_p + x];
            float diffus_sq = diffus*diffus;
            while(true)
            {
                float change = 0;
                for(unsigned int i1 = 0; i1 < nr_labels; i1++)
                    for(unsigned int i2 = i1; i2 < nr_labels; i2++)
                    {
                        //compute sum over all input images between i1 and i2 for each component x,y
                        float v1 = 0;
                        float v2 = 0;
                        for(unsigned int j = i1; j <= i2; j++)
                        {
                            v1 += dual[pos     + page*j];
                            v2 += dual[pos + c + page*j];
                        }
                        float mm = v1*v1+v2*v2;  //compute length of the summarized vector
                        if(mm > diffus_sq)
                        {
                            mm = sqrt(mm);
                            //remove the length above 1 (-> m - 1) in direction of normalized sum vector avg
                            float rest = (mm - diffus)/(i2 - i1 + 1);
                            float mod1 = rest * (v1 / mm);
                            float mod2 = rest * (v2 / mm);
                            for(unsigned int j = i1; j <= i2; j++)
                            {
                                dual[pos     + page*j] -= mod1;
                                dual[pos + c + page*j] -= mod2;
                                change += mod1*mod1+mod2*mod2;
                            }
                        }
                    }
                if (change < 0.01f) break;
            }

        }

    }

    __syncthreads();

    // 2nd and 3rd step
    if (work)
    {


        // Add tau*(div-likely) to primal and check boundaries
        for(int l=0; l < nr_labels;l++)
        {
            int off =l*c + pos;
            int tl_off  = l*lt_p*ny + y*lt_p+x;
            float div = 0;

            if (x>WIN_SIZE)
                div -= dual[l*page     + pos-1] ;
            if (x<nx-1-WIN_SIZE)
                div += dual[l*page     +pos];
            if (y>WIN_SIZE)
                div -= dual[l*page + c + pos-pd_p];
            if (y<ny-1-WIN_SIZE)
                div += dual[l*page + c +pos];

            tmp[tl_off] = primal[off];
            primal[off] += tau*(div-likely[tl_off]);

        }

        // Simplex projection
        bool finished=false;
        while(!finished)
        {
            finished=true;
            // Determine n (nonzero-dimension of vector) and the sum of the entries
            int n = 0;
            float sum=0.0f;
            for(int l=0; l < nr_labels;l++)
            {
                float value = primal[c*l + pos];
                if(value!=0) n++;
                sum += value;
            }


            // Do projection
            for(int l=0; l < nr_labels;l++)
            {
                float value = primal[c*l + pos];
                if(value!=0) value -= (sum-1.0f)/(float)n;
                if(value < 0.0f)
                {
                    value = 0.0f;
                    finished = false;
                }
                primal[c*l + pos] = value;
                primal[book + c*l + pos] = value;
            }
        }



        // Acceleration step
        for(int l=0; l < nr_labels;l++)
        {
            int off =l*c + pos;
            int tl_off  = l*lt_p*ny + y*lt_p+x;
            primal[off] =  2*primal[off] - tmp[tl_off];
        }


    }

    __syncthreads();


}

bool gpu_segmentation(GPU_DATA data, GPU_DATA const_gpu,int &currIt)
{

    int nx = data.int_params[NX];
    int ny = data.int_params[NY];
    int nr_labels = data.int_params[NR_LABELS];
    int nr_seg_it = data.int_params[NR_SEG_IT];
    float tau = data.float_params[SEG_TAU];

    float size = 16;
    dim3 blockSize(size, size);
    dim3 gridSize( (int)ceil(nx/size), (int)ceil(ny/size) );


    CUDA(hipMemcpy2D((void*) const_gpu.likely,   const_gpu.lt_p,data.likely,nx*sizeof(float) , nx*sizeof(float), ny*nr_labels,hipMemcpyHostToDevice),"MemCpyLikely");
    CUDA(hipMemcpy2D((void*) const_gpu.g,        const_gpu.g_p,data.g,     nx*sizeof(float) , nx*sizeof(float), ny,hipMemcpyHostToDevice),"MemCpyG");

    if(currIt==0)
    {
        CUDA(hipMemset2D((void*)const_gpu.primal, const_gpu.pd_p,0, nx*sizeof(float), ny*nr_labels*2), "MemsetPrimal") ;
        CUDA(hipMemset2D((void*)const_gpu.dual,   const_gpu.pd_p,0, nx*sizeof(float), ny*nr_labels*2), "MemsetDual") ;
    }
    else
    {
        CUDA(hipMemcpy2D((void*) const_gpu.primal, const_gpu.pd_p,data.primal,nx*sizeof(float) , nx*sizeof(float), 2*ny*nr_labels,hipMemcpyHostToDevice),"MemCpyPrimal");
        CUDA(hipMemcpy2D((void*) const_gpu.dual,   const_gpu.pd_p,data.dual  ,nx*sizeof(float) , nx*sizeof(float), 2*ny*nr_labels,hipMemcpyHostToDevice),"MemCpyDual");
    }


    int dual_space = 0;
    if(data.stepwise)
    {

        for(int i=0; i < nr_seg_it;i++,currIt++)
        {
            KERNEL (kernel_segmentation,gridSize, blockSize)
                    (const_gpu.likely, const_gpu.primal,const_gpu.dual,const_gpu.temp,const_gpu.g,tau,nx,ny,nr_labels,
                     const_gpu.lt_p/sizeof(float),const_gpu.pd_p/sizeof(float),const_gpu.g_p/sizeof(float),dual_space);
        }

    }
    else
    {

        int start_it = currIt;
        while(currIt++-start_it<1500 )
        {

            KERNEL (kernel_segmentation,gridSize, blockSize)
                    (const_gpu.likely, const_gpu.primal,const_gpu.dual,const_gpu.temp,const_gpu.g,tau,nx,ny,nr_labels,
                     const_gpu.lt_p/sizeof(float),const_gpu.pd_p/sizeof(float),const_gpu.g_p/sizeof(float),dual_space);
        }


    }

    CUDA(hipDeviceSynchronize(),"Syncsegmentation");
    CUDA(hipMemcpy2D((void*) data.primal,  nx*sizeof(float),const_gpu.primal, const_gpu.pd_p, nx*sizeof(float), ny*nr_labels,hipMemcpyDeviceToHost),"MemCpyPrimal");
    CUDA(hipMemcpy2D((void*) data.dual  ,  nx*sizeof(float),const_gpu.dual,   const_gpu.pd_p, nx*sizeof(float), 2*ny*nr_labels,hipMemcpyDeviceToHost),"MemCpyDual");


    return true;


}


__device__ float kernel_gauss(float x,float var)
{
    return expf(-0.5f*x*x/(var*var))/(var*SQRT_2PI);
}

__device__ float p2NormSq(int x1, int y1, int x2, int y2)
{
    int a = (x1-x2);
    int b = (y1-y2);
    return a*a+b*b;
}


__global__ void kernel_density(int *scribbles, float *colors, float *textures, float *params, float *likely,float *temp,
                               int nx, int ny, int nr_labels, int nr_scribbles,int tex_dim, int *label_count,
                               size_t like_p,size_t col_p,size_t tex_p)
{

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    int border = 0;
    if (x >= border && x < nx-border && y >= border  && y < ny-border) {

        float alpha = params[0];
        //float delta = params[1];
        float scale = 1.0f/max(nx,ny);
        int tex_page = tex_p*ny;
        int col_page = col_p*ny;

        // Find NN-scribble
        if(alpha>0)
        {
            float nnScribble_dist= nx*nx+ny*ny;
            for(int i=0; i < nr_scribbles; i++)
            {
                int sx = scribbles[i*3+0];
                int sy = scribbles[i*3+1];
                float dist = p2NormSq(x,y,sx,sy);
                if(dist < nnScribble_dist)
                    nnScribble_dist=dist;
            }
            if(nnScribble_dist < 1.f) nnScribble_dist=1.f;
            alpha*= sqrt(nnScribble_dist)*scale;
        }

        // Run through scribbles and estimate
        for(int i=0; i < nr_scribbles; i++)
        {
            float space=1.0f,color=1.0f,texture=1.0f;
            int sx = scribbles[i*3+0];
            int sy = scribbles[i*3+1];
            int slabel = scribbles[i*3+2];
            float sigma = params[2+slabel];
            float beta =  params[2+nr_labels+slabel];

            if(alpha>0)
            {
                float v1 = x-sx;
                float v2 = y-sy;
                float distance = v1*(v1*temp[i*3+0] + v2*temp[i*3+2]) +
                        v2*(v1*temp[i*3+2] + v2*temp[i*3+1]);
                space = kernel_gauss(sqrt(distance)*scale,alpha);
            }

            if(sigma>0)
            {
                float r = colors[y*col_p+x]                - colors[sy*col_p+sx];
                float g = colors[y*col_p+x + col_page]     - colors[sy*col_p+sx + col_page];
                float b = colors[y*col_p+x + 2*col_page]   - colors[sy*col_p+sx + 2*col_page];

                color = kernel_gauss(r,sigma)*kernel_gauss(g,sigma)*kernel_gauss(b,sigma);
            }

            if(beta>0)
            {
                /*
               float diff=0;
               for(int t = 0; t < tex_dim; t++)
               {
                   int bits = ((int)textures[t*tex_page+y*tex_p+x]) ^ ((int) textures[t*tex_page+sy*tex_p+sx]);
                   while (bits)
                     {
                         diff++;
                         bits &= bits - 1;
                    }
               }
                texture = kernel_gauss(diff,beta);
                */

                for(int t = 0; t < tex_dim; t++)
                {
                    float diff = textures[t*tex_page+y*tex_p+x]-textures[t*tex_page+sy*tex_p+sx];
                    texture *= kernel_gauss(diff,beta);
                }

            }

            likely[ny*like_p*slabel + y*like_p + x] += space*color*texture;

        }

        // Divide by scribble number
        for(int i=0; i < nr_labels;i++)
            likely[ny*like_p*i + y*like_p + x] /= ((float)label_count[i]);

    }

}


bool gpu_density(GPU_DATA data, GPU_DATA const_gpu)
{



    int nx = data.int_params[NX];
    int ny = data.int_params[NY];
    int nr_labels = data.int_params[NR_LABELS];
    int nr_scribbles = data.int_params[NR_SCRIBBLES];
    int tex_dim = data.int_params[TEX_DIM];
    float size = 16;
    dim3 blockSize(size, size);
    dim3 gridSize( (int)ceil(nx/size), (int)ceil(ny/size) );


    CUDA(hipMemset2D((void*)const_gpu.likely, const_gpu.lt_p,0, nx*sizeof(float), ny*nr_labels), "MemsetLikely") ;

    CUDA(hipMemcpy(const_gpu.scribbles, data.scribbles, nr_scribbles*sizeof(int)*3,hipMemcpyHostToDevice),"MemCopyScribbles");
    CUDA(hipMemcpy2D((void*) const_gpu.colors, const_gpu.col_p,data.colors,nx*sizeof(float) , nx*sizeof(float), ny*3,hipMemcpyHostToDevice),"MemCpyColors");
    CUDA(hipMemcpy2D((void*) const_gpu.textures, const_gpu.tex_p,data.textures,nx*sizeof(float) , nx*sizeof(float), ny*tex_dim,hipMemcpyHostToDevice),"MemCpyTex");
    CUDA(hipMemcpy(const_gpu.label_count, data.label_count, nr_labels*sizeof(int),hipMemcpyHostToDevice),"MemCopyLabelCount");
    CUDA(hipMemcpy(const_gpu.float_params, data.float_params,   (2+2*nr_labels)*sizeof(float),hipMemcpyHostToDevice),"MemCopyFLOATPARAMS");
    CUDA(hipMemcpy(const_gpu.temp, data.temp,   nr_scribbles*3*sizeof(float),hipMemcpyHostToDevice),"MemCopyTemp");

    KERNEL(kernel_density,gridSize,blockSize)(const_gpu.scribbles, const_gpu.colors, const_gpu.textures,const_gpu.float_params,
                                              const_gpu.likely, const_gpu.temp,
                                              nx,ny,nr_labels,nr_scribbles,tex_dim, const_gpu.label_count,
                                              const_gpu.lt_p/sizeof(float),const_gpu.col_p/sizeof(float),const_gpu.tex_p/sizeof(float));


    CUDA(hipMemcpy2D((void*) data.likely, nx*sizeof(float),const_gpu.likely, const_gpu.lt_p, nx*sizeof(float), ny*nr_labels,hipMemcpyDeviceToHost),"MemCpyLikely");

    return true;


}


__global__ void kernel_wavelet(float *hh, float *hl, float *lh,
                               float *hh_avg, float *hl_avg, float *lh_avg,
                               float *hh_stddev, float *hl_stddev, float *lh_stddev,
                               int nx, int ny, int win_size, int nr_wav_steps, size_t pitch )
{

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    int startx = max(x-win_size,0), endx = min(x+win_size,nx);
    int starty = max(y-win_size,0), endy = min(y+win_size,ny);
    if (x > win_size && x < nx-win_size-1 && y > win_size  && y < ny-win_size-1) {


        for(int n=0; n < nr_wav_steps;n++)
        {
            int off = n*pitch*ny;
            int nrVals=0;
            float mean_hh=0.0f, mean_hl=0.0f, mean_lh=0.0f;
            for(int i= startx; i < endx; i++)
                for(int j= starty; j < endy; j++)
                {
                    int pos = off+j*pitch+i;
                    mean_hh += (hh[pos]);
                    mean_hl += (hl[pos]);
                    mean_lh += (lh[pos]);
                    nrVals++;
                }

            hh_avg[off+y*pitch+x] =  mean_hh/nrVals;
            hl_avg[off+y*pitch+x] =  mean_hl/nrVals;
            lh_avg[off+y*pitch+x] =  mean_lh/nrVals;
        }
    }

    __syncthreads();

    if (x >= win_size && x < nx-win_size && y >= win_size  && y < ny-win_size) {
        for(int n=0; n < nr_wav_steps;n++)
        {
            int off = n*pitch*ny;
            int nrVals=0;
            float mean_hh=0.0f, mean_hl=0.0f, mean_lh=0.0f;
            for(int i= startx; i < endx; i++)
                for(int j= starty; j < endy; j++)
                {
                    int pos = off+j*pitch+i;
                    float res = (hh[pos]) - hh_avg[pos];
                    mean_hh += res*res;
                    res = (hl[pos]) - hl_avg[pos];
                    mean_hl += res*res;
                    res = (lh[pos]) - lh_avg[pos];
                    mean_lh += res*res;
                    nrVals++;
                }

            hh_stddev[off+y*pitch+x] =  sqrtf(mean_hh/(nrVals));
            hl_stddev[off+y*pitch+x] =  sqrtf(mean_hl/(nrVals));
            lh_stddev[off+y*pitch+x] =  sqrtf(mean_lh/(nrVals));
        }
    }

}



bool gpu_wavelet(GPU_DATA data)
{

    int nx = data.int_params[NX];
    int ny = data.int_params[NY];
    int win_size = data.int_params[WAVELET_WIN_SIZE];
    int nr_wav_steps = data.int_params[WAVELET_STEPS];

    float size = 8;
    dim3 blockSize(size, size);
    dim3 gridSize( (int)ceil(nx/size), (int)ceil(ny/size) );

    float *hh,*hl,*lh,*hh_avg,*hl_avg,*lh_avg,*hh_stddev,*hl_stddev,*lh_stddev;
    size_t pitch;


    CUDA(hipMallocPitch((void**) &hh,&pitch,  nx*sizeof(float), ny*nr_wav_steps), "MallocHH") ;
    CUDA(hipMallocPitch((void**) &hl,&pitch,  nx*sizeof(float), ny*nr_wav_steps), "MallocHL") ;
    CUDA(hipMallocPitch((void**) &lh,&pitch,  nx*sizeof(float), ny*nr_wav_steps), "MallocLH") ;

    CUDA(hipMemcpy2D((void*) hh, pitch,data.hh,nx*sizeof(float) , nx*sizeof(float), ny*nr_wav_steps,hipMemcpyHostToDevice),"MemCpyHH");
    CUDA(hipMemcpy2D((void*) hl, pitch,data.hl,nx*sizeof(float) , nx*sizeof(float), ny*nr_wav_steps,hipMemcpyHostToDevice),"MemCpyHL");
    CUDA(hipMemcpy2D((void*) lh, pitch,data.lh,nx*sizeof(float) , nx*sizeof(float), ny*nr_wav_steps,hipMemcpyHostToDevice),"MemCpyLH");


    CUDA(hipMallocPitch((void**) &hh_avg,&pitch,  nx*sizeof(float), ny*nr_wav_steps), "MallocHHAVG") ;
    CUDA(hipMallocPitch((void**) &hl_avg,&pitch,  nx*sizeof(float), ny*nr_wav_steps), "MallocHLAVG") ;
    CUDA(hipMallocPitch((void**) &lh_avg,&pitch,  nx*sizeof(float), ny*nr_wav_steps), "MallocLHAVG") ;
    CUDA(hipMallocPitch((void**) &hh_stddev,&pitch,  nx*sizeof(float), ny*nr_wav_steps), "MallocHHSTDDEV") ;
    CUDA(hipMallocPitch((void**) &hl_stddev,&pitch,  nx*sizeof(float), ny*nr_wav_steps), "MallocHLSTDDEV") ;
    CUDA(hipMallocPitch((void**) &lh_stddev,&pitch,  nx*sizeof(float), ny*nr_wav_steps), "MallocLHSTDDEV") ;

    CUDA(hipMemset2D((void*)hh_avg, pitch,0, nx*sizeof(float), ny*nr_wav_steps), "MemsetHHAVG") ;
    CUDA(hipMemset2D((void*)hl_avg, pitch,0, nx*sizeof(float), ny*nr_wav_steps), "MemsetHLAVG") ;
    CUDA(hipMemset2D((void*)lh_avg, pitch,0, nx*sizeof(float), ny*nr_wav_steps), "MemsetLHAVG") ;
    CUDA(hipMemset2D((void*)hh_stddev, pitch,0, nx*sizeof(float), ny*nr_wav_steps), "MemsetHHSTDDEV") ;
    CUDA(hipMemset2D((void*)hl_stddev, pitch,0, nx*sizeof(float), ny*nr_wav_steps), "MemsetHLSTDDEV") ;
    CUDA(hipMemset2D((void*)lh_stddev, pitch,0, nx*sizeof(float), ny*nr_wav_steps), "MemsetLHSTDDEV") ;


    KERNEL(kernel_wavelet,gridSize,blockSize)(hh,hl,lh,hh_avg,hl_avg,lh_avg,hh_stddev,hl_stddev,lh_stddev,
                                              nx,ny,win_size,nr_wav_steps,pitch/sizeof(float));


    CUDA(hipMemcpy2D((void*) data.hh_avg, nx*sizeof(float),hh_avg, pitch, nx*sizeof(float), ny*nr_wav_steps,hipMemcpyDeviceToHost),"MemCpyHH_AVG");
    CUDA(hipMemcpy2D((void*) data.hl_avg, nx*sizeof(float),hl_avg, pitch, nx*sizeof(float), ny*nr_wav_steps,hipMemcpyDeviceToHost),"MemCpyHL_AVG");
    CUDA(hipMemcpy2D((void*) data.lh_avg, nx*sizeof(float),lh_avg, pitch, nx*sizeof(float), ny*nr_wav_steps,hipMemcpyDeviceToHost),"MemCpyLH_AVG");
    CUDA(hipMemcpy2D((void*) data.hh_stddev, nx*sizeof(float),hh_stddev, pitch, nx*sizeof(float), ny*nr_wav_steps,hipMemcpyDeviceToHost),"MemCpyHH_STDDEV");
    CUDA(hipMemcpy2D((void*) data.hl_stddev, nx*sizeof(float),hl_stddev, pitch, nx*sizeof(float), ny*nr_wav_steps,hipMemcpyDeviceToHost),"MemCpyHL_STDDEV");
    CUDA(hipMemcpy2D((void*) data.lh_stddev, nx*sizeof(float),lh_stddev, pitch, nx*sizeof(float), ny*nr_wav_steps,hipMemcpyDeviceToHost),"MemCpyLH_STDDEV");


    hipFree(lh);
    hipFree(hh);
    hipFree(hl);
    hipFree(hh_avg);
    hipFree(hl_avg);
    hipFree(lh_avg);
    hipFree(hh_stddev);
    hipFree(hl_stddev);
    hipFree(lh_stddev);

    return true;


}

bool gpu_wavelet(GPU_DATA data, GPU_DATA const_gpu)
{

    int nx = data.int_params[NX];
    int ny = data.int_params[NY];
    int win_size = data.int_params[WAVELET_WIN_SIZE];
    int nr_wav_steps = data.int_params[WAVELET_STEPS];

    float size = 8;
    dim3 blockSize(size, size);
    dim3 gridSize( (int)ceil(nx/size), (int)ceil(ny/size) );





    CUDA(hipMemcpy2D((void*) const_gpu.hh, const_gpu.wave_p,data.hh,nx*sizeof(float) , nx*sizeof(float), ny*nr_wav_steps,hipMemcpyHostToDevice),"MemCpyHH");
    CUDA(hipMemcpy2D((void*) const_gpu.hl, const_gpu.wave_p,data.hl,nx*sizeof(float) , nx*sizeof(float), ny*nr_wav_steps,hipMemcpyHostToDevice),"MemCpyHL");
    CUDA(hipMemcpy2D((void*) const_gpu.lh, const_gpu.wave_p,data.lh,nx*sizeof(float) , nx*sizeof(float), ny*nr_wav_steps,hipMemcpyHostToDevice),"MemCpyLH");


    CUDA(hipMemset2D((void*)const_gpu.hh_avg, const_gpu.wave_p,0, nx*sizeof(float), ny*nr_wav_steps), "MemsetHHAVG") ;
    CUDA(hipMemset2D((void*)const_gpu.hl_avg, const_gpu.wave_p,0, nx*sizeof(float), ny*nr_wav_steps), "MemsetHLAVG") ;
    CUDA(hipMemset2D((void*)const_gpu.lh_avg, const_gpu.wave_p,0, nx*sizeof(float), ny*nr_wav_steps), "MemsetLHAVG") ;
    CUDA(hipMemset2D((void*)const_gpu.hh_stddev, const_gpu.wave_p,0, nx*sizeof(float), ny*nr_wav_steps), "MemsetHHSTDDEV") ;
    CUDA(hipMemset2D((void*)const_gpu.hl_stddev, const_gpu.wave_p,0, nx*sizeof(float), ny*nr_wav_steps), "MemsetHLSTDDEV") ;
    CUDA(hipMemset2D((void*)const_gpu.lh_stddev, const_gpu.wave_p,0, nx*sizeof(float), ny*nr_wav_steps), "MemsetLHSTDDEV") ;


    KERNEL(kernel_wavelet,gridSize,blockSize)(const_gpu.hh,const_gpu.hl,const_gpu.lh,const_gpu.hh_avg,const_gpu.hl_avg,const_gpu.lh_avg,
                                              const_gpu.hh_stddev,const_gpu.hl_stddev,const_gpu.lh_stddev,
                                              nx,ny,win_size,nr_wav_steps,const_gpu.wave_p/sizeof(float));


    CUDA(hipMemcpy2D((void*) data.hh_avg, nx*sizeof(float),const_gpu.hh_avg, const_gpu.wave_p, nx*sizeof(float), ny*nr_wav_steps,hipMemcpyDeviceToHost),"MemCpyHH_AVG");
    CUDA(hipMemcpy2D((void*) data.hl_avg, nx*sizeof(float),const_gpu.hl_avg, const_gpu.wave_p, nx*sizeof(float), ny*nr_wav_steps,hipMemcpyDeviceToHost),"MemCpyHL_AVG");
    CUDA(hipMemcpy2D((void*) data.lh_avg, nx*sizeof(float),const_gpu.lh_avg, const_gpu.wave_p, nx*sizeof(float), ny*nr_wav_steps,hipMemcpyDeviceToHost),"MemCpyLH_AVG");
    CUDA(hipMemcpy2D((void*) data.hh_stddev, nx*sizeof(float),const_gpu.hh_stddev, const_gpu.wave_p, nx*sizeof(float), ny*nr_wav_steps,hipMemcpyDeviceToHost),"MemCpyHH_STDDEV");
    CUDA(hipMemcpy2D((void*) data.hl_stddev, nx*sizeof(float),const_gpu.hl_stddev, const_gpu.wave_p, nx*sizeof(float), ny*nr_wav_steps,hipMemcpyDeviceToHost),"MemCpyHL_STDDEV");
    CUDA(hipMemcpy2D((void*) data.lh_stddev, nx*sizeof(float),const_gpu.lh_stddev, const_gpu.wave_p, nx*sizeof(float), ny*nr_wav_steps,hipMemcpyDeviceToHost),"MemCpyLH_STDDEV");



    return true;


}


